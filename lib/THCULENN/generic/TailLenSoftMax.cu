
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/TailLenSoftMax.cu"
#else

#include "../common.h"

void THLENN_(TailLenSoftMax_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *len)
{
  THCULENN_assertSameGPU(state, 2, input, output);

  if ((input->nDimension != 2) && (len->nDimension != 1))
  {
    THError("2D tensor expected for input, 1D tensor expected for len");
  }

  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, output, input);
  long batchSize = input->size[0], dim = input->size[1];
  long blocksY = 1, blocksZ = 1;

  dim3 blocks(batchSize, blocksY, blocksZ);
  dim3 threads(TAILLENSOFTMAX_THREADS);
  culenn_TailLenSoftMax_updateOutput_kernel<real, accreal, THCIndex_t><<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, output),
    THCTensor_(data)(state, input),
    batchSize, dim, THCIndexTensor_(data)(state, len)
  );
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
}

void THLENN_(TailLenSoftMax_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           THCIndexTensor *len)
{
  THCULENN_check_nElement(state, input, gradOutput);
  THCULENN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  if ((gradInput->nDimension != 2) && (len->nDimension != 1))
  {
    THError("2D tensor expected for input, 1D tensor expected for len");
  }


  output = THCTensor_(newContiguous)(state, output);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, output);
  long batchSize = gradInput->size[0], dim = gradInput->size[1];
  long blocksY = 1, blocksZ = 1;

  dim3 blocks(batchSize, blocksY, blocksZ);
  dim3 threads(TAILLENSOFTMAX_THREADS);
  culenn_TailLenSoftMax_updateGradInput_kernel<real, accreal, THCIndex_t><<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, gradInput),
    THCTensor_(data)(state, output),
    THCTensor_(data)(state, gradOutput),
    batchSize, dim, THCIndexTensor_(data)(state, len)
  );
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, output);
}

#endif
